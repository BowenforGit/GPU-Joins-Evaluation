#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include <iostream>
#include <vector>
#include <tuple>
#include <chrono>
#include <unistd.h>
#include <fstream>

#include <hip/hip_runtime.h>

#include "tuple.cuh"
#include "utils.cuh"
#include "operators.cuh"
#include "tpc_utils.hpp"
#include "../data_gen/generator.cuh"
#include "sort_merge_join.cuh"
#include "partitioned_hash_join.cuh"
#include "sort_hash_join.cuh"
#include "experiment_util.cuh"
#include "join_base.hpp"

using namespace std;

#define PAYLOAD_COL_T long
#define KEY_COL_T long

#define LOAD_Q64_COL(dir, p, N, from, to, seed) \
    to* p; \
    GET_DATA_TYPE(p) \
    { \
        std::string filename = dir + "/" + #p + ".bin"; \
        read_col<from,to>(filename, p, (N), true, (seed)); \
    }

#define LOAD_Q64_COL_ALL(dir, Nss, Ncd) \
    LOAD_Q64_COL((dir), ss_cdemo_sk, (Nss), int, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_customer_sk, (Nss), long, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_addr_sk, (Nss), long, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_item_sk, (Nss), long, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_ticket_number, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_wholesale_cost, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_list_price, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_coupon_amt, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), d_year, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), s_store_name, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), s_zip, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), cd_demo_sk, (Ncd), int, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), cd_marital_status, (Ncd), long, PAYLOAD_COL_T, 42)

void run_q64(std::string algo, std::string profile_output_) {
    const int Nss = 57898426;
    const int Ncd = 1920800;
    std::string dir(TPC_DATA_PREFIX"tpcds_sf100/q64");

    LOAD_Q64_COL_ALL(dir, Nss, Ncd)
  
    using ss_t = struct Chunk<ss_cdemo_sk_t,ss_customer_sk_t,ss_addr_sk_t,ss_item_sk_t,ss_ticket_number_t,ss_wholesale_cost_t,ss_list_price_t,ss_coupon_amt_t,d_year_t,s_store_name_t,s_zip_t>;
    using cd_t = struct Chunk<cd_demo_sk_t, cd_marital_status_t>;
    
    using join_t = struct Chunk<cd_demo_sk_t,cd_marital_status_t,ss_customer_sk_t,ss_addr_sk_t,ss_item_sk_t,ss_ticket_number_t,ss_wholesale_cost_t,ss_list_price_t,ss_coupon_amt_t,d_year_t,s_store_name_t,s_zip_t>;

    auto cd_cols = std::make_tuple(cd_demo_sk, cd_marital_status);
    auto ss_cols = std::make_tuple(ss_cdemo_sk,ss_customer_sk,ss_addr_sk,ss_item_sk,ss_ticket_number,ss_wholesale_cost,ss_list_price,ss_coupon_amt,d_year,s_store_name,s_zip);

    ScanOperator<cd_t> cd_scan(std::move(cd_cols), Ncd, Ncd);
    ScanOperator<ss_t> ss_scan(std::move(ss_cols), Nss, Nss);

    cd_scan.open(); ss_scan.open();
    auto relation_cd = cd_scan.next();
    auto relation_ss = ss_scan.next();
    cd_scan.close(); ss_scan.close();

    const int circular_buffer_size = Nss;
    const int first_bit = 0;
    const int log_part1 = 9;
    const int log_part2 = 6;
    
    JoinBase<join_t> *impl;
    if (algo == "SMJ") {
        impl = new SortMergeJoin<cd_t, ss_t, join_t, true>(relation_cd, relation_ss, circular_buffer_size);
    } else if (algo == "PHJ") {
        impl = new PartitionHashJoin<cd_t, ss_t, join_t>(relation_cd, relation_ss, log_part1, log_part2, first_bit, circular_buffer_size);
    } else if (algo == "SHJ") {
        impl = new SortHashJoin<cd_t, ss_t, join_t>(relation_cd, relation_ss, first_bit, log_part1+log_part2, circular_buffer_size);
    } else if (algo == "SMJI") {
        impl = new SortMergeJoinByIndex<cd_t, ss_t, join_t>(relation_cd, relation_ss, circular_buffer_size);
    } 
    else {
        std::cout << "Invalid algorithm name: " << algo << std::endl;
        exit(1);
    }

    auto out = impl->join();
    hipDeviceSynchronize();

    cout << "\nOutput Cardinality = " << out.num_items << endl;
    cout << "Results (first 10 items): \n";
    out.peek(min(10, out.num_items));

    impl->print_stats();

    std::ofstream fout;
    fout.open(profile_output_, ios::app);
    fout << get_utc_time() << ","
        << relation_cd.num_items << "," << relation_ss.num_items << ","
        << algo << ",";

    auto stats = impl->all_stats();
    for(auto t : stats) {
        fout << t << ",";
    }

    fout << std::endl;
    fout.close();

    relation_cd.free_mem();
    relation_ss.free_mem();
    out.free_mem();
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cout << "Usage: ./bin/volcano/q64 <algo> <profile_output>" << std::endl;
        exit(1);
    }
    std::string algo = argv[1];
    std::string profile_output = argv[2];
    run_q64(algo, profile_output);
    return 0;
}